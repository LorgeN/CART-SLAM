#include "hip/hip_runtime.h"
#include <opencv2/core/cuda_stream_accessor.hpp>

#include "cartslam.hpp"
#include "modules/disparity.hpp"
#include "modules/disparity/interpolation.cuh"
#include "utils/cuda.cuh"

#define THREADS_PER_BLOCK_X 16
#define THREADS_PER_BLOCK_Y 16
#define X_BATCH 4
#define Y_BATCH 4

#define SHARED_SIZE(radius) (((X_BATCH * THREADS_PER_BLOCK_X + (radius - 1) * 2) * (Y_BATCH * THREADS_PER_BLOCK_Y + (radius - 1) * 2)) * sizeof(cart::disparity_t))

#define LOCAL_INDEX(x, y) SHARED_INDEX(sharedPixelX + x, sharedPixelY + y, radius - 1, radius - 1, sharedRowStep)

__global__ void interpolateKernel(cv::cuda::PtrStepSz<cart::disparity_t> disparity, int radius, int width, int height, int iterations, int minDisparity, int maxDisparity) {
    extern __shared__ cart::disparity_t shared[];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int sharedPixelX = threadIdx.x * X_BATCH;
    int sharedPixelY = threadIdx.y * Y_BATCH;

    int pixelX = x * X_BATCH;
    int pixelY = y * Y_BATCH;

    size_t sharedRowStep = X_BATCH * blockDim.x;

    cart::copyToShared<cart::disparity_t, X_BATCH, Y_BATCH>(shared, disparity, radius - 1, radius - 1);

    const unsigned int minCount = radius * radius + 1;  // Requires about 1 fourth of the pixels to be valid

    __syncthreads();

    // Average neighboring pixels
    for (int i = 0; i < iterations; i++) {
        for (int j = 0; j < X_BATCH; j++) {
            for (int i = 0; i < Y_BATCH; i++) {
                if (pixelX + j >= width || pixelY + i >= height) {
                    continue;
                }

                int sum = 0;
                int count = 0;

                for (int k = -radius + 1; k < radius; k++) {
                    for (int l = -radius + 1; l < radius; l++) {
                        cart::disparity_t value = shared[LOCAL_INDEX(j + k, i + l)];

                        if (value > minDisparity && value < maxDisparity) {
                            sum += value;
                            count++;
                        }
                    }
                }

                if (count > minCount) {
                    shared[LOCAL_INDEX(j, i)] = sum / count;
                } else {
                    shared[LOCAL_INDEX(j, i)] = CARTSLAM_DISPARITY_INVALID;
                }
            }
        }

        __syncthreads();
    }

    size_t disparityStep = disparity.step / sizeof(cart::disparity_t);

    // Write back to global memory
    for (int i = 0; i < Y_BATCH; i++) {
        for (int j = 0; j < X_BATCH; j++) {
            if (pixelX + j >= width || pixelY + i >= height) {
                continue;
            }

            disparity[INDEX(pixelX + j, pixelY + i, disparityStep)] = shared[LOCAL_INDEX(j, i)];
        }
    }
}

namespace cart::disparity {
void interpolate(log4cxx::LoggerPtr logger, cv::cuda::GpuMat& disparity, cv::cuda::Stream& stream, int radius, int iterations, int minDisparity, int maxDisparity) {
    int width = disparity.cols;
    int height = disparity.rows;

    dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
    dim3 numBlocks((disparity.cols + (threadsPerBlock.x * X_BATCH - 1)) / (threadsPerBlock.x * X_BATCH),
                   (disparity.rows + (threadsPerBlock.y * Y_BATCH - 1)) / (threadsPerBlock.y * Y_BATCH));

    int sharedSize = SHARED_SIZE(radius);

    hipStream_t cudaStream = cv::cuda::StreamAccessor::getStream(stream);
    interpolateKernel<<<numBlocks, threadsPerBlock, sharedSize, cudaStream>>>(disparity, radius, width, height, iterations, minDisparity, maxDisparity);

    CUDA_SAFE_CALL(logger, hipGetLastError());
}
}  // namespace cart::disparity