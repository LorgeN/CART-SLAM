#include "hip/hip_runtime.h"
#include <opencv2/cudaimgproc.hpp>

#include "modules/disparity.hpp"
#include "modules/superpixels.hpp"
#include "modules/superpixels/contourrelaxation/features/color.cuh"
#include "modules/superpixels/contourrelaxation/features/compactness.cuh"
#include "modules/superpixels/contourrelaxation/features/disparity.cuh"
#include "modules/superpixels/visualization.cuh"
#include "utils/modules.hpp"

namespace cart {
SuperPixelModule::SuperPixelModule(
    const unsigned int initialIterations,
    const unsigned int blockWidth,
    const unsigned int blockHeight,
    const double directCliqueCost,
    const double compactnessWeight)
    : SuperPixelModule::SyncWrapperSystemModule("SuperPixelDetect", {CARTSLAM_KEY_DISPARITY_DERIVATIVE}),
      initialIterations(initialIterations) {
    if (blockWidth < 1 || blockHeight < 1) {
        throw std::invalid_argument("blockWidth and blockHeight must be more than 1");
    }

    if (directCliqueCost < 0) {
        throw std::invalid_argument("directCliqueCost must be non-negative");
    }

    if (compactnessWeight < 0) {
        throw std::invalid_argument("compactnessWeight must be non-negative");
    }

    cv::cuda::GpuMat initialLabelImage;
    cart::contour::label_t maxLabelId;
    contour::createBlockInitialization(cv::Size(CARTSLAM_IMAGE_RES_X, CARTSLAM_IMAGE_RES_Y), blockWidth, blockHeight, initialLabelImage, maxLabelId);

    this->contourRelaxation = boost::make_shared<contour::ContourRelaxation>(initialLabelImage, maxLabelId, directCliqueCost, directCliqueCost / sqrt(2));
    this->contourRelaxation->addFeature<contour::CompactnessFeature>(compactnessWeight);

#ifdef CARTSLAM_IMAGE_MAKE_GRAYSCALE
    this->contourRelaxation->addFeature<contour::DisparityFeature>(0.6);
    this->contourRelaxation->addFeature<contour::GrayvalueFeature>(0.75);
#else
    this->contourRelaxation->addFeature<contour::DisparityFeature>(1.5);
    this->contourRelaxation->addFeature<contour::ColorFeature>(0.75);
#endif
}

system_data_t SuperPixelModule::runInternal(System &system, SystemRunData &data) {
    cv::cuda::GpuMat image;
    cv::cuda::Stream stream;

#ifdef CARTSLAM_IMAGE_MAKE_GRAYSCALE
    // Generate a 3-channel version of the grayscale image, which we will need later on
    // to generate the boundary overlay. Save it in the "image" variable so we won't
    // have to care about the original type of the image anymore.
    cv::cuda::cvtColor(getReferenceImage(data.dataElement), image, cv::COLOR_GRAY2BGR, 0, stream);
#else
    // Convert image to YUV-like YCrCb for uncorrelated color channels.
    cv::cuda::cvtColor(getReferenceImage(data.dataElement), image, cv::COLOR_BGR2YCrCb, 0, stream);
#endif

    stream.waitForCompletion();

    auto disparityDerivative = data.getData<cv::cuda::GpuMat>(CARTSLAM_KEY_DISPARITY_DERIVATIVE);

    const unsigned int numIterations = data.id == 1 ? this->initialIterations : 8;

    cv::cuda::GpuMat relaxedLabelImage;

    {
        // Lock the mutex to protect the contour relaxation object, which is not thread safe
        boost::lock_guard<boost::mutex> lock(this->mutex);
        this->contourRelaxation->relax(numIterations, image, *disparityDerivative, relaxedLabelImage);
    }

    return MODULE_RETURN_SHARED(CARTSLAM_KEY_SUPERPIXELS, cv::cuda::GpuMat, relaxedLabelImage);
}

boost::future<system_data_t> SuperPixelVisualizationModule::run(System &system, SystemRunData &data) {
    auto promise = boost::make_shared<boost::promise<system_data_t>>();

    boost::asio::post(system.getThreadPool(), [this, promise, &system, &data]() {
        auto labels = data.getData<cv::cuda::GpuMat>(CARTSLAM_KEY_SUPERPIXELS);

        cv::cuda::GpuMat image = getReferenceImage(data.dataElement);
        cv::cuda::GpuMat boundaryOverlay;

        cart::contour::computeBoundaryOverlay(this->logger, image, *labels, boundaryOverlay);

        cv::Mat boundaryOverlayCpu;
        boundaryOverlay.download(boundaryOverlayCpu);

        this->imageThread->setImageIfLater(boundaryOverlayCpu, data.id);
        promise->set_value(MODULE_NO_RETURN_VALUE);
    });

    return promise->get_future();
}
}  // namespace cart