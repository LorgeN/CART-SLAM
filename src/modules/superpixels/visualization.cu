#include "hip/hip_runtime.h"
#include "modules/superpixels/visualization.cuh"
#include "utils/cuda.cuh"

#define THREADS_PER_BLOCK_X 32
#define THREADS_PER_BLOCK_Y 32
#define X_BATCH 4
#define Y_BATCH 4

__global__ void overlayBoundaryVisualization(cv::cuda::PtrStepSz<uint8_t> bgrImage, cv::cuda::PtrStepSz<cart::contour::label_t> labels, cv::cuda::PtrStepSz<uint8_t> out) {
    int x = (blockIdx.x * blockDim.x + threadIdx.x) * X_BATCH;
    int y = (blockIdx.y * blockDim.y + threadIdx.y) * Y_BATCH;

    size_t bgrStep = bgrImage.step / sizeof(uint8_t);
    size_t labelsStep = labels.step / sizeof(cart::contour::label_t);
    size_t outStep = out.step / sizeof(uint8_t);

    for (int i = 0; i < Y_BATCH; i++) {
        for (int j = 0; j < X_BATCH; j++) {
            if (x + j >= labels.cols - 1 || y + i >= labels.rows - 1) {
                continue;
            }

            uint8_t b = bgrImage[INDEX_BGR(x + j, y + i, 0, bgrStep)];
            uint8_t g = bgrImage[INDEX_BGR(x + j, y + i, 1, bgrStep)];
            uint8_t r = bgrImage[INDEX_BGR(x + j, y + i, 2, bgrStep)];

            cart::contour::label_t label = labels[INDEX(x + j, y + i, labelsStep)];
            cart::contour::label_t right = labels[INDEX(x + j + 1, y + i, labelsStep)];
            cart::contour::label_t down = labels[INDEX(x + j, y + i + 1, labelsStep)];

            if (label != right || label != down) {
                b = 0;
                g = 0;
                r = 255;
            }

            out[INDEX_BGR(x + j, y + i, 0, outStep)] = b;
            out[INDEX_BGR(x + j, y + i, 1, outStep)] = g;
            out[INDEX_BGR(x + j, y + i, 2, outStep)] = r;
        }
    }
}

namespace cart::contour {

void computeBoundaryOverlay(log4cxx::LoggerPtr logger, cv::cuda::GpuMat bgrImage, cv::cuda::GpuMat labelImage, cv::cuda::GpuMat &out_boundaryOverlay) {
    assert(bgrImage.type() == CV_8UC3);
    assert(labelImage.type() == cv::DataType<label_t>::type);

    if (out_boundaryOverlay.empty() || out_boundaryOverlay.size() != bgrImage.size()) {
        out_boundaryOverlay.create(bgrImage.size(), CV_8UC3);
    }

    dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
    dim3 numBlocks((labelImage.cols + threadsPerBlock.x * X_BATCH - 1) / (threadsPerBlock.x * X_BATCH), (labelImage.rows + threadsPerBlock.y * Y_BATCH - 1) / (threadsPerBlock.y * Y_BATCH));

    hipStream_t stream;
    CUDA_SAFE_CALL(logger, hipStreamCreate(&stream));

    overlayBoundaryVisualization<<<numBlocks, threadsPerBlock, 0, stream>>>(bgrImage, labelImage, out_boundaryOverlay);

    CUDA_SAFE_CALL(logger, hipGetLastError());
    CUDA_SAFE_CALL(logger, hipStreamSynchronize(stream));
    CUDA_SAFE_CALL(logger, hipStreamDestroy(stream));
}
}  // namespace cart::contour